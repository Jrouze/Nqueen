#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <sys/time.h>
#define _QUEENS_BLOCK_SIZE_ 	128
#define _EMPTY_      -1
#define _MAX_DEPTH_ 12
#define _MAX_QUEEN_SIZE_ 32

double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}

//keeps the feasible, valid and incomplete solutions
typedef struct queen_root{
    unsigned int control;
    int8_t board[_MAX_DEPTH_];
} QueenRoot;


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


//this is to get the feasible-valid and incomplete solution into the data structure
inline void prefixesHandleSol(QueenRoot *root_prefixes,unsigned int flag,char *board,int initialDepth,int num_sol){

    root_prefixes[num_sol].control = flag;

    for(int i = 0; i<initialDepth;++i)
      root_prefixes[num_sol].board[i] = (char)board[i];
}

//verifies if the configuration is legal -- CPU
inline bool MCstillLegal(const char *board, const int r)
{

    int i;
    int ld;
    int rd;
  // Check vertical
  for ( i = 0; i < r; ++i)
    if (board[i] == board[r]) return false;
    // Check diagonals
    ld = board[r];  //left diagonal columns
    rd = board[r];  // right diagonal columns
    for ( i = r-1; i >= 0; --i) {
      --ld; ++rd;
      if (board[i] == ld || board[i] == rd) return false;
    }

    return true;
}

//verifies if the configuration is legal -- GPU (Not the same as on CPU)
__device__  bool GPU_queens_stillLegal(const char *board, const int r){

  bool safe = true;
  int i;
  register int ld;
  register int rd;
  // Check vertical
  for ( i = 0; i < r; ++i)
    if (board[i] == board[r]) safe = false;
    // Check diagonals
    ld = board[r];  //left diagonal columns
    rd = board[r];  // right diagonal columns
    for ( i = r-1; i >= 0; --i) {
      --ld; ++rd;
      if (board[i] == ld || board[i] == rd) safe = false;
    }

    return safe;
}

//the GPU Kernel
__global__ void BP_queens_root_dfs(int N, unsigned int nPreFixos, int depthPreFixos,
    QueenRoot *root_prefixes,unsigned long long int *vector_of_tree_size, unsigned long long int *sols){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nPreFixos) {
        unsigned int flag = 0;
        unsigned int bit_test = 0;
        char board[_MAX_QUEEN_SIZE_]; //keeps the solution
        int N_l = N;
        int i, depth;
        unsigned long long  qtd_solucoes_thread = 0ULL;
        int depthGlobal = depthPreFixos;
        unsigned long long int tree_size = 0ULL;

        for (i = 0; i < N_l; ++i) {
            board[i] = _EMPTY_;
        }

        flag = root_prefixes[idx].control;


        for (i = 0; i < depthGlobal; ++i)
            board[i] = root_prefixes[idx].board[i];

        depth=depthGlobal;

        do{

            board[depth]++;
            bit_test = 0;
            bit_test |= (1<<board[depth]);

            if(board[depth] == N_l){
                board[depth] = _EMPTY_;
                //if(block_ub > upper)   block_ub = upper;
            }else if (!(flag &  bit_test ) && GPU_queens_stillLegal(board, depth)){


                    flag |= (1ULL<<board[depth]);
                    tree_size++;
                    depth++;

                    if (depth == N_l) { //sol
                        ++qtd_solucoes_thread;
                    }else continue;
                }else continue;

            depth--;
            flag &= ~(1ULL<<board[depth]);

            }while(depth >= depthGlobal); //FIM DO DFS_BNB

        sols[idx] = qtd_solucoes_thread;
        vector_of_tree_size[idx] = tree_size;
    }//if
}//kernel
////////


//the partial seach on CPU -- finds feasible, valid and incomplete solutions
unsigned long long int BP_queens_prefixes(int size, int initialDepth ,unsigned long long *tree_size, QueenRoot *root_prefixes){

    unsigned int flag = 0;
    int bit_test = 0;
    char board[_MAX_QUEEN_SIZE_]; //the board wich keeps the solution at hand
    int i, depth; //the initial depth of the search
    unsigned long long int local_tree = 0ULL;
    unsigned long long int num_sol = 0;
   //register int custo = 0;

    /*initialization*/
    for (i = 0; i < size; ++i) { //
        board[i] = -1;
    }

    depth = 0;

    do{

        board[depth]++;
        bit_test = 0;
        bit_test |= (1<<board[depth]);


        if(board[depth] == size){
            board[depth] = _EMPTY_;
                //if(block_ub > upper)   block_ub = upper;
        }else if ( MCstillLegal(board, depth) && !(flag &  bit_test ) ){ //is legal

                flag |= (1ULL<<board[depth]);
                depth++;
                ++local_tree;
                if (depth == initialDepth){ //handle solution
                   prefixesHandleSol(root_prefixes,flag,board,initialDepth,num_sol);
                   num_sol++;
            }else continue;
        }else continue;

        depth--;
        flag &= ~(1ULL<<board[depth]);

    }while(depth >= 0);

    *tree_size = local_tree;

    return num_sol;
}


//CUDA memory manipulation and calling both searches

void GPU_call_cuda_queens(int size, int initial_depth, int block_size, bool set_cache, unsigned int n_explorers, QueenRoot *root_prefixes_h ,
	unsigned long long int *vector_of_tree_size_h, unsigned long long int *sols_h, int gpu_id){

    hipSetDevice(gpu_id);

    if(set_cache){
        printf("\n ### nSeeting up the cache ###\n");
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(BP_queens_root_dfs),hipFuncCachePreferL1);
    }


    unsigned long long int *vector_of_tree_size_d;
    unsigned long long int *sols_d;
    QueenRoot *root_prefixes_d;

    int num_blocks = ceil((double)n_explorers/block_size);


    hipMalloc((void**) &vector_of_tree_size_d,n_explorers*sizeof(unsigned long long int));
    hipMalloc((void**) &sols_d,n_explorers*sizeof(unsigned long long int));
    hipMalloc((void**) &root_prefixes_d,n_explorers*sizeof(QueenRoot));

    //I Think this is not possible in Chapel. It must be internal
    hipMemcpy(root_prefixes_d, root_prefixes_h, n_explorers * sizeof(QueenRoot), hipMemcpyHostToDevice);

    printf("\n### Regular BP-DFS search. ###\n");

    //kernel_start =  rtclock();

    BP_queens_root_dfs<<< num_blocks,block_size>>> (size,n_explorers,initial_depth,root_prefixes_d, vector_of_tree_size_d,sols_d);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    //kernel_stop = rtclock();

    hipMemcpy(vector_of_tree_size_h,vector_of_tree_size_d,n_explorers*sizeof(unsigned long long int),hipMemcpyDeviceToHost);
    hipMemcpy(sols_h,sols_d,n_explorers*sizeof(unsigned long long int),hipMemcpyDeviceToHost);

    hipFree(vector_of_tree_size_d);
    hipFree(sols_d);
    hipFree(root_prefixes_d);

}

double call_queens(int size, int initialDepth, int block_size, int set_cache){


    unsigned long long initial_tree_size = 0ULL;
    unsigned long long qtd_sols_global = 0ULL;
    unsigned long long gpu_tree_size = 0ULL;


    unsigned int nMaxPrefixos = 75580635;

    printf("\n### Queens size: %d, Initial depth: %d, Block size: %d", initialDepth, size, block_size);
    double initial_time = rtclock();

    QueenRoot* root_prefixes_h = (QueenRoot*)malloc(sizeof(QueenRoot)*nMaxPrefixos);
    unsigned long long int *vector_of_tree_size_h = (unsigned long long int*)malloc(sizeof(unsigned long long int)*nMaxPrefixos);
    unsigned long long int *solutions_h = (unsigned long long int*)malloc(sizeof(unsigned long long int)*nMaxPrefixos);

    //initial search, getting the tree root nodes for the gpu;
    unsigned long long n_explorers = BP_queens_prefixes((short)size, initialDepth ,&initial_tree_size, root_prefixes_h);

    //calling the gpu-based search

    GPU_call_cuda_queens(size, initialDepth, block_size, (bool)set_cache,n_explorers, root_prefixes_h ,vector_of_tree_size_h, solutions_h, 0);

    printf("\nInitial tree size: %llu", initial_tree_size );

    double final_time = rtclock();

    for(int i = 0; i<n_explorers;++i){
        if(solutions_h[i]>0)
            qtd_sols_global += solutions_h[i];
        if(vector_of_tree_size_h[i]>0)
            gpu_tree_size += vector_of_tree_size_h[i];
    }


    printf("\nGPU Tree size: %llu\nTotal tree size: %llu\nNumber of solutions found: %llu.\n", gpu_tree_size,(initial_tree_size+gpu_tree_size),qtd_sols_global );
    printf("\nElapsed total: %.3f\n", (final_time-initial_time));

    return (final_time-initial_time);
}


int main(int argc, char *argv[]){

    int initialDepth;
    int size;
    int block_size;

    block_size = atoi(argv[3]);
    initialDepth = atoi(argv[2]);
    size = atoi(argv[1]);

    auto time=call_queens(size, initialDepth, block_size,0);

    FILE *f;
    f=fopen("data_single_cuda.txt","a");
    fprintf(f, "%d %d %d %f \n", size, initialDepth, block_size, time );

    return 0;
}
